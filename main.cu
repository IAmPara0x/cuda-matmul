#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "matrix.h"
#include "matmul.h"

using namespace std;

#define THREADS 32

int main(void) {

    getDeviceInfo();

    Matrix A = readMat("A.txt");
    Matrix B = readMat("B.txt");
    Matrix C = readMat("C.txt");
    Matrix result = alloc_mat(A.rows, A.cols);

    cout << "Matrix Size: " << A.rows << "x" << A.cols << endl;

    float *cuMat1 = nullptr, *cuMat2 = nullptr, *cuMatResult = nullptr;

    hipMalloc((void **)&cuMat1, A.size);
    hipMalloc((void **)&cuMat2, B.size);
    hipMalloc((void **)&cuMatResult, A.size);

    // cuMemcpy()
    hipMemcpy(cuMat1, A.value, A.size, hipMemcpyHostToDevice);
    hipMemcpy(cuMat2, B.value, A.size, hipMemcpyHostToDevice);


    // NOTE:: START TIMER

    auto start = std::chrono::high_resolution_clock::now();
    
    dim3 threadsPerBlock(THREADS, THREADS);
    dim3 blocks((A.rows + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (A.rows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatMulKernel<<<blocks, threadsPerBlock>>>(cuMat1, cuMat2, cuMatResult, A.rows);
    hipDeviceSynchronize();

    // NOTE:: END TIMER
    auto end = std::chrono::high_resolution_clock::now();

    hipMemcpy(result.value, cuMatResult, A.size, hipMemcpyDeviceToHost);

    // hipFree
    hipFree(cuMat1);
    hipFree(cuMat2);
    hipFree(cuMatResult);

    std::chrono::duration<double> duration = end - start;

    if (result == C) 
        cout << "Test Passed!" << endl;
    else
        cout << "Test Failed!" << endl;

    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    // host free
    free_mat(A);
    free_mat(B);
    free_mat(C);
    free_mat(result);
    return 0;
}

void getDeviceInfo() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    std::cout << "Device Name: " << props.name << std::endl;
    std::cout << "Number of SMs: " << props.multiProcessorCount << std::endl;
    std::cout << "Max Threads per SM: " << props.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Max Threads per Block: " << props.maxThreadsPerBlock << std::endl;
    std::cout << "Shared Memory per Block: " << props.sharedMemPerBlock << std::endl;
    return;
};
