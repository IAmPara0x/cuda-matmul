#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hipblas.h>
#include "matrix.h"
#include "matmul.h"

using namespace std;


void square_matmul(float *A, float *B, float *C, size_t N) {

    // NOTE:: START TIMER
    auto start = std::chrono::high_resolution_clock::now();

    dim3 threadsPerBlock(THREADS, THREADS);
    dim3 blocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatMulKernel<<<blocks, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();

    // NOTE:: END TIMER
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;
}

void cublas_matmul(hipblasHandle_t handle, float *A, float *B, float *C, size_t N) {

    // Initialize cuBLAS context

    // Parameters for matrix multiplication
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // NOTE:: START TIMER

    auto start = std::chrono::high_resolution_clock::now();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A, N, B, N, &beta, C, N);

    // NOTE:: END TIMER
    auto end = std::chrono::high_resolution_clock::now();

}


int main(void) {

    getDeviceInfo();

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    Matrix A = readMat("A.txt");
    Matrix B = readMat("B.txt");
    Matrix C = readMat("C.txt");
    Matrix result = alloc_mat(A.rows, A.cols);

    transpose(&B);
    
    cout << "Matrix Size: " << A.rows << "x" << A.cols << endl;

    float *cuMat1 = nullptr, *cuMat2 = nullptr, *cuMatResult = nullptr;

    hipMalloc((void **)&cuMat1, A.size);
    hipMalloc((void **)&cuMat2, B.size);
    hipMalloc((void **)&cuMatResult, A.size);

    // cuMemcpy()
    hipMemcpy(cuMat1, A.value, A.size, hipMemcpyHostToDevice);
    hipMemcpy(cuMat2, B.value, A.size, hipMemcpyHostToDevice);

    square_matmul(cuMat1, cuMat2, cuMatResult, A.rows);
    // cublas_matmul(handle, cuMat1, cuMat2, cuMatResult, A.rows);

    hipMemcpy(result.value, cuMatResult, A.size, hipMemcpyDeviceToHost);

    // hipFree
    hipFree(cuMat1);
    hipFree(cuMat2);
    hipFree(cuMatResult);
    hipblasDestroy(handle);

    if (result == C) 
        cout << "Test Passed!" << endl;
    else
        cout << "Test Failed!" << endl;


    // host free
    free_mat(A);
    free_mat(B);
    free_mat(C);
    free_mat(result);
    return 0;
}


void getDeviceInfo() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    std::cout << "Device Name: " << props.name << std::endl;
    std::cout << "Number of SMs: " << props.multiProcessorCount << std::endl;
    std::cout << "Max Threads per SM: " << props.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Max Threads per Block: " << props.maxThreadsPerBlock << std::endl;
    std::cout << "Shared Memory per Block: " << props.sharedMemPerBlock << std::endl;
    return;
};
