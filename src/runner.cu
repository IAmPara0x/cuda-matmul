#include "functional"
#include "matmul.h"
#include "matrix.h"
#include "runner.h"

using namespace std;


#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck1(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

#define cudaCheck(err) (cudaCheck1(err, __FILE__, __LINE__))

function<void()> runner(hipblasHandle_t handle, MatMulKernel kernel,
                             HostMatrices host, DeviceMatrices device,
                             size_t N) {

  if (kernel == MatMulKernelCuBLAS) {
    return ([handle, device, N]() {
      cuBlas_MatMul(handle, device.dA, device.dB, device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });
  }

  dim3 blockDim(THREADS, THREADS);
  dim3 gridDim(CEIL_DIV(N, THREADS), CEIL_DIV(N, THREADS));

  if (kernel == MatMulKernelNaive) {
    return ([handle, device, N, blockDim, gridDim]() {
      MatMulKernel_Naive<<<gridDim, blockDim>>>(device.dA, device.dB, device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });
  }

  hipMemcpy(device.dB, host.hB, matrix_size(host.hB, N),
             hipMemcpyHostToDevice);

  std::function<void()> func;

  if (kernel == MatMulKernelStrided)
    func = ([handle, device, N, blockDim, gridDim]() {
      MatMulKernel_Strided<<<gridDim, blockDim>>>(device.dA, device.dB,
                                                        device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });

  return func;
}

std::string matmulKernelToString(MatMulKernel kernel) {

  switch (kernel) {
  case MatMulKernel::MatMulKernelCuBLAS:
    return "MatMulKernelCuBLAS";
  case MatMulKernel::MatMulKernelNaive:
    return "MatMulKernelNaive";
  case MatMulKernel::MatMulKernelStrided:
    return "MatMulKernelStrided";
  default:
    return "UNKNOWN";
  }
}
