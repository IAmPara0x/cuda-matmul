#include "functional"
#include "matmul.h"
#include "matrix.h"
#include "runner.h"

using namespace std;


void cudaCheck1(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

#define cudaCheck(err) (cudaCheck1(err, __FILE__, __LINE__))

function<void()> runner(hipblasHandle_t handle, MatMulKernel kernel,
                             HostMatrices host, DeviceMatrices device,
                             size_t N) {

  if (kernel == MatMulKernelCuBLAS) {
    return ([handle, device, N]() {
      cuBlas_MatMul(handle, device.dA, device.dB, device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });
  }

  dim3 threadsPerBlock(THREADS, THREADS);
  dim3 blocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
              (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  if (kernel == MatMulKernelNaive) {
    return ([handle, device, N, threadsPerBlock, blocks]() {
      MatMulKernel_Naive<<<blocks, threadsPerBlock>>>(device.dA, device.dB, device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });
  }

  transpose(host.hB, N);
  hipMemcpy(device.dB, host.hB, matrix_size(host.hB, N),
             hipMemcpyHostToDevice);

  std::function<void()> func;

  if (kernel == MatMulKernelRowMajor)
    func = ([handle, device, N, threadsPerBlock, blocks]() {
      MatMulKernel_RowMajor<<<blocks, threadsPerBlock>>>(device.dA, device.dB,
                                                         device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });

  if (kernel == MatMulKernelStrided)
    func = ([handle, device, N, threadsPerBlock, blocks]() {
      MatMulKernel_Strided<<<blocks, threadsPerBlock>>>(device.dA, device.dB,
                                                        device.dC, N);
      cudaCheck(hipDeviceSynchronize());
    });

  transpose(host.hB, N);
  return func;
}

std::string matmulKernelToString(MatMulKernel kernel) {

  switch (kernel) {
  case MatMulKernel::MatMulKernelCuBLAS:
    return "MatMulKernelCuBLAS";
  case MatMulKernel::MatMulKernelNaive:
    return "MatMulKernelNaive";
  case MatMulKernel::MatMulKernelRowMajor:
    return "MatMulKernelRowMajor";
  case MatMulKernel::MatMulKernelStrided:
    return "MatMulKernelStrided";
  default:
    return "UNKNOWN";
  }
}
